#include "hip/hip_runtime.h"
#include "cu_utils.hh"


__global__ void setup_curand_rng(const int seed, hiprandState *state, const int gpulen)
{
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx >= gpulen)
    return;
  
  hiprand_init(seed, idx, 0, state + idx);
}
