#include "hip/hip_runtime.h"
#include "include/cu_utils.hh"
#include "include/rand_utils.hh"
#include "include/Rcurand.h"


__global__ void rpois(hiprandState *state, const double lambda, const double ignore, const int gpulen, int *x)
{
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx >= gpulen)
    return;
  
  x[idx] = hiprand_poisson(state + idx, lambda);
}



extern "C" SEXP R_curand_poisson(SEXP n1_, SEXP n2_, SEXP lambda_, SEXP seed_)
{
  SEXP x;
  const int32_t n1 = INT(n1_);
  const int32_t n2 = INT(n2_);
  const R_xlen_t n = (R_xlen_t)n1*n1 + n2;
  
  const unsigned int seed = INTEGER(seed_)[0];
  const double lambda = REAL(lambda_)[0];
  
  
  PROTECT(x = allocVector(INTSXP, n));
  curand_rng_driver(seed, n, lambda, 0.0, INTEGER(x), rpois);
  
  
  UNPROTECT(1);
  return x;
}
